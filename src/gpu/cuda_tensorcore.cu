#include <mma.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include <cmath>
using namespace nvcuda;
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

__global__ void wmma_gemm(const half *A, const half *B, float *C, int N) {
    int tile_i = blockIdx.y * WMMA_M;
    int tile_j = blockIdx.x * WMMA_N;
    if (tile_i + WMMA_M <= N && tile_j + WMMA_N <= N) {
        wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> b_frag;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
        wmma::fill_fragment(c_frag, 0.0f);
        for (int k = 0; k < N; k += WMMA_K) {
            const half *tile_a = A + (tile_i*N + k);
            const half *tile_b = B + (k*N + tile_j);
            wmma::load_matrix_sync(a_frag, tile_a, N);
            wmma::load_matrix_sync(b_frag, tile_b, N);
            wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
        }
        wmma::store_matrix_sync(C + tile_i*N + tile_j, c_frag, N, wmma::mem_row_major);
    }
}

// In header: extern void cuda_tensorcore_multiply(const float*, const float*, float*, int);
// API wrapper
void to_half(const float* src, half* dst, int size) {
    for (int i = 0; i < size; ++i) dst[i] = __float2half(src[i]);
}

void cuda_tensorcore_multiply(const float* h_A, const float* h_B, float* h_C, int N) {
    size_t size = N * N;
    size_t bytes_A = size * sizeof(half);
    size_t bytes_B = size * sizeof(half);
    size_t bytes_C = size * sizeof(float);

    std::vector<half> h_A_half(size), h_B_half(size);
    to_half(h_A, h_A_half.data(), size);
    to_half(h_B, h_B_half.data(), size);

    half *d_A, *d_B;
    float *d_C;
    hipMalloc(&d_A, bytes_A); hipMalloc(&d_B, bytes_B); hipMalloc(&d_C, bytes_C);
    hipMemcpy(d_A, h_A_half.data(), bytes_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B_half.data(), bytes_B, hipMemcpyHostToDevice);
    hipMemset(d_C, 0, bytes_C);

    dim3 grid(N / WMMA_M, N / WMMA_N); dim3 block(32, 1);
    wmma_gemm<<<grid, block>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    hipMemcpy(h_C, d_C, bytes_C, hipMemcpyDeviceToHost);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}
