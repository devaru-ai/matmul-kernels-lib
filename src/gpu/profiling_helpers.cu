#include <hip/hip_runtime.h>
#include <iostream>
float profile_kernel(void (*kernel)(const float*, const float*, float*, int),
    const float* h_A, const float* h_B, float* h_C, int N) {
    hipEvent_t start, stop; hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);
    kernel(h_A, h_B, h_C, N);
    hipEventRecord(stop); hipEventSynchronize(stop);
    float ms=0.0f;
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start); hipEventDestroy(stop);
    return ms;
}
// Example usage: float ms=profile_kernel(cuda_naive_multiply, h_A, h_B, h_C, N);
